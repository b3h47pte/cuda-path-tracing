#include "hip/hip_runtime.h"
#include "cuda_path_integrator.h"
#include "gpgpu/cuda_acceleration_structure.h"
#include "gpgpu/cuda_intersection.h"
#include "gpgpu/cuda_utils.h"
#include "gpgpu/math/cuda_vector.h"

namespace cpt {
namespace {

__global__ void path_trace(CudaRay* rays, const CudaScene* scene, size_t sampleIdx, CudaAovOutput* output) {
    const int pixelIdx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t x, y;
    output->get_xy_from_flat_index(x, y, pixelIdx);

    CudaRay& ray = rays[pixelIdx];

    // See if we intersect anything in the scene.
    CudaIntersection intersection;
    if (!ray_geometry_intersect(&ray, scene->accel_structure(), &intersection)) {
        ray.set_alive(false);
        return;
    }

    CudaVector<float, 3> rgb;
    rgb[intersection.hit_geometry->id() % 3] = 1.f;

    // Output.
    for (size_t imgIdx = 0; imgIdx < output->num_images(); ++imgIdx) {
        AovOutput::Channels channel = output->channel(imgIdx);
        CudaImage* img = output->image(imgIdx);
        if (channel == AovOutput::Channels::FinalImage) {
            img->accumulate(
                rgb / static_cast<float>(sampleIdx + 1),
                x,
                y, 
                static_cast<float>(sampleIdx) / (sampleIdx + 1));
        }
    }
}

}

void CudaPathIntegrator::Li(CudaRay* rays, size_t num_rays, const CudaScene* scene, size_t sampleIdx, CudaAovOutput* output) const {
    int blocks, threads;
    compute_blocks_threads(blocks, threads, num_rays);
    path_trace<<<blocks, threads>>>(rays, scene, sampleIdx, output);
}

}
