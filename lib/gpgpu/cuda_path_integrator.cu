#include "hip/hip_runtime.h"
#include "cuda_path_integrator.h"
#include "gpgpu/math/cuda_vector.h"
#include "gpgpu/cuda_utils.h"

namespace cpt {
namespace {

__global__ void path_trace(CudaRay* rays, const CudaScene* scene, size_t sampleIdx, CudaAovOutput* output) {
    const int pixelIdx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t x, y;
    output->get_xy_from_flat_index(x, y, pixelIdx);

    CudaVector<float, 3> rgb;
    rgb[pixelIdx % 3] = 1.f;

    // Output.
    for (size_t imgIdx = 0; imgIdx < output->num_images(); ++imgIdx) {
        AovOutput::Channels channel = output->channel(imgIdx);
        CudaImage* img = output->image(imgIdx);
        if (channel == AovOutput::Channels::FinalImage) {
            img->accumulate(
                rgb / static_cast<float>(sampleIdx + 1),
                x,
                y, 
                static_cast<float>(sampleIdx) / (sampleIdx + 1));
        }
    }
}

}

void CudaPathIntegrator::Li(CudaRay* rays, size_t num_rays, const CudaScene* scene, size_t sampleIdx, CudaAovOutput* output) const {
    int blocks, threads;
    compute_blocks_threads(blocks, threads, num_rays);
    path_trace<<<blocks, threads>>>(rays, scene, sampleIdx, output);
}

}
