#include "cuda_sampler.h"
#include "gpgpu/cuda_utils.h"

namespace cpt {

CUDA_DEVICE CudaSampler::CudaSampler(unsigned long long seed) {
    generate_curand_state(seed); 
}

CUDA_DEVICE CudaSampler::~CudaSampler() {
}

CUDA_DEVICE void CudaSampler::generate_curand_state(unsigned long long seed) {
    hiprand_init(seed, get_cuda_flat_thread_index(), 0, &_curand_state);
}

}
